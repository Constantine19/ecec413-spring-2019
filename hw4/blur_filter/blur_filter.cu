#include "hip/hip_runtime.h"
/* Reference code implementing the box blur filter.

  Build and execute as follows: 
    make clean && make 
    ./blur_filter size

  Author: Naga Kandasamy
  Date created: May 3, 2019
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

/* #define DEBUG */

/* Include the kernel code. */
#include "blur_filter_kernel.cu"

extern "C" void compute_gold (const image_t, image_t);
void compute_on_device (const image_t, image_t);
int check_results (const float *, const float *, int, float);
void print_image (const image_t);
void check_CUDA_error (const char *);

int 
main (int argc, char **argv)
{
    if (argc < 2) {
        printf ("Usage: %s size\n", argv[0]);
        printf ("size: Height of the image. The program assumes size x size image.\n");
        exit (EXIT_FAILURE);
    }

    /* Allocate memory for the input and output images. */
    int size = atoi (argv[1]);

    printf ("Creating %d x %d images\n", size, size);
    image_t in, out_gold, out_gpu;
    in.size = out_gold.size = out_gpu.size = size;
    in.element = (float *) malloc (sizeof (float) * size * size);
    out_gold.element = (float *) malloc (sizeof (float) * size * size);
    out_gpu.element = (float *) malloc (sizeof (float) * size * size);
    if ((in.element == NULL) || (out_gold.element == NULL) || (out_gpu.element == NULL)) {
        perror ("Malloc");
        exit (EXIT_FAILURE);
    }

    /* Poplulate our image with random values between [-0.5 +0.5] */
    srand (time (NULL));
    for (int i = 0; i < size * size; i++)
        in.element[i] = rand ()/ (float) RAND_MAX -  0.5;
        // in.element[i] = 1;
  
   /* Calculate the blur on the CPU. The result is stored in out_gold. */
    struct timeval start, stop;
    printf ("Calculating blur on the CPU\n");
    gettimeofday(&start, NULL);
   compute_gold (in, out_gold);
    gettimeofday(&stop, NULL);
    printf ("Serial time: %0.2f seconds\n", (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) / (float) 1000000) );
#ifdef DEBUG 
   print_image (in);
   print_image (out_gold);
#endif

   /* Calculate the blur on the GPU. The result is stored in out_gpu. */
   printf ("Calculating blur on the GPU\n");
   compute_on_device (in, out_gpu);

   /* Check the CPU and GPU results for correctness. */
   printf ("Checking CPU and GPU results\n");
   int num_elements = out_gold.size * out_gold.size;
   float eps = 1e-6;
   int check = check_results (out_gold.element, out_gpu.element, num_elements, eps);
   if (check == 1) 
       printf ("TEST PASSED\n");
   else
       printf ("TEST FAILED\n");
   
   /* Free data structures on the host. */
   free ((void *) in.element);
   free ((void *) out_gold.element);
   free ((void *) out_gpu.element);

    exit (EXIT_SUCCESS);
}

/* FIXME: Complete this function to calculate the blur on the GPU. */
void 
compute_on_device (const image_t in, image_t out)
{
    float *din = in.element;
    float *dout;
    int n = in.size;
    int size = sizeof (float) * (n * n);

    // Allocate memory on device for input array
    hipMalloc ((void**) &din, size);
    check_CUDA_error ("Error allocating memory");

    // Copy data from host to device
    hipMemcpy (din, in.element, size, hipMemcpyHostToDevice);
    check_CUDA_error ("Error copying data from host to device");

    // Allocate memory on device for output array
    hipMalloc ((void**) &dout, size);
    check_CUDA_error ("Error allocating memory");

    dim3 threads (32, 32); // Threads per block
    dim3 grid (n/threads.x, n/threads.y);
    struct timeval start, stop;
    gettimeofday(&start, NULL);
    blur_filter_kernel<<<grid, threads>>>(din, dout, n);
    gettimeofday(&stop, NULL);
    printf ("Parallel time: %f seconds\n", (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) / (float) 1000000) );
    check_CUDA_error ("Error in kernel");

    hipMemcpy (out.element, dout, sizeof (float) * (n * n), hipMemcpyDeviceToHost);
    check_CUDA_error ("Error copying result from device");

    hipFree(din);
    hipFree(dout);
    check_CUDA_error ("Error freeing memory");
 
    return;
}

void
check_CUDA_error (const char *msg)
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) {
       printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
       exit (EXIT_FAILURE);
    }
}

/* Function to check correctness of the results. */
int 
check_results (const float *pix1, const float *pix2, int num_elements, float eps) 
{
    for (int i = 0; i < num_elements; i++)
        if (fabsf ((pix1[i] - pix2[i])/pix1[i]) > eps) {
            printf("%f        %f -- %d", pix1[i], pix2[i], i);
            return 0;
        }

    return 1;
}

/* Function to print out the image contents. */
void 
print_image (const image_t img)
{
    for (int i = 0; i < img.size; i++) {
        for (int j = 0; j < img.size; j++) {
            float val = img.element[i * img.size + j];
            printf ("%0.4f ", val);
        }
        printf ("\n");
    }

    printf ("\n");
    return;
}
